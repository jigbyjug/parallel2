#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vectorAdd(float *g_odata, float *g_idata, int n)
{
	extern __shared__ float temp[]; //allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;

	temp[2*thid] = g_idata[2*thid];		// load input into shared memory
	temp[2*thid+1] = g_idata[2*thid+1];
		
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree 
	{
		__syncthreads();
		
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;

			temp[bi] += temp[1i];
		}
		offset *= 2;
	}
		

	if (thid==0)  { temp[n-1] = 0;} // clear the last element

	for (int d = 1; d < n; d*= 2)   // traverse down tree & build scan
	{
		offset >>=1;
		__syncthreads();

		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads(); 

	g_odata[2*thid] = temp[2*thid]; //write results to device memory
	g_odata[2*thid+1] = temp[2*thid+1];
}


int main(void) 
{
	int numElements = 15; 
	size_t size = numElements * sizeof(float);
	
	//Allocate the host input vector A
	float *h_A = (float *)malloc(size);
	
	//Verify that allocations succeeded
	if (h_A == Null)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}
	
	//Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() % 101;
		cout<<h_A[i]<<'\n';
	}	

	// Allocate the deivce input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector A!\n");
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector B
    	float *d_B = NULL;
    	err = hipMalloc((void **)&d_B, size);
	
	if (err != hipSuccess)
        {
                fprintf(stderr, "Failed to allocate device vector B!\n");
                exit(EXIT_FAILURE);
        }


	//Copy the host input vectors A in host memory to the device input vectors
	// in device memory

	 printf("Copy input data from the host memory to the CUDA device\n");
	 err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

   	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector A from host to device!\n");
        	exit(EXIT_FAILURE);
    	}


	printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
                fprintf(stderr, "Failed to copy vector A from host to device!\n");
                exit(EXIT_FAILURE);
        }





	// Launch the vector add cuda kernel
	int threadsPerBlock = 256;	
	int blocksPerGrid = (numElements + threadsPerBlock - 1)/threadsPerBlock;
	
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements);


	//Copy the device result vector in device memory to the host result vector
	// in host memory

	err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector B from device to host!\n");
        	exit(EXIT_FAILURE);
    	}

	for (int i = 0; i < numElements; i++)
	{
		cout<<h_B[i]<<'\n';
	}


	// Free device global memory
	err = hipFree(d_A);

	if (err != hipSuccess)
        {
                fprintf(stderr, "Failed to free device vector A!\n");
                exit(EXIT_FAILURE);
        }

	err = hipFree(d_B);

	if (err != hipSuccess)
        {
                fprintf(stderr, "Failed to free device vector B!\n");
                exit(EXIT_FAILURE);
        }

	//free host memory
	free(h_A);
	free(h_B);

	// reset device and exit
	err = hipDeviceReset();
	
	if (err != hipSuccess)
        {
                fprintf(stderr, "Failed to deinitialize the device!\n");
                exit(EXIT_FAILURE);
        }

	return 0;
}



